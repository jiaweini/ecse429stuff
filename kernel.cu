#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "lodepng.c"
#include ""

#include <iostream>
#include <stdio.h>
#include <algorithm>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);



__global__ void addKernel(unsigned char* image, unsigned char* image_out, int threads,
	int width, int height, int width_output,int height_output, int pix_per_thread) {

	int thread_index = blockIdx.x * 1024+ threadIdx.x;
	int starting_pixel = thread_index * pix_per_thread; // start at (including) this pixel
	int ending_pixel = thread_index == threads - 1 ? width_output * height_output : starting_pixel + pix_per_thread; //stop before this pixel

	for (int pix = starting_pixel; pix < ending_pixel; pix+=4) {
		int index_input = (width*4) + (pix % width_output)*4 + (pix/width_output*width*4); // corresponding block's top left pixel(input) *4(channel)
		int index_output = pix * 4;
		image_out[index_output] = image[index_input];
		image_out[index_output +1] = image[index_input +1];
		image_out[index_output +2] = image[index_input +2];
		image_out[index_output +3] = image[index_input +3];
	
	}


}



void loadPNG(char* input_filename, unsigned int& width, unsigned int& height, unsigned char** output_data)
{
	unsigned error;

	error = lodepng_decode32_file(output_data, &width, &height, input_filename);
	if (error) printf("error %u: %s\n", error, lodepng_error_text(error));
}

void exportPNG(char* output_filename, unsigned int width, unsigned int height,
	unsigned char* data)
{
	lodepng_encode32_file(output_filename, data, width, height);
}



void convolution(char* input_filename, char* output_filename,int dimension, int threads)
{
	unsigned char* image = nullptr;
	unsigned char* new_image=nullptr;
	unsigned int width, height;
	loadPNG(input_filename, width, height, &image);
	int size_input = width * height * 4;  // input size (channels)
	int width_output = width - dimension + 1; // output width (pixels)
	int height_output = height - dimension + 1; //output height (pixels)
	int pixels_output = width_output * height_output;
	int size_output = width_output * height_output * 4; // output size (channels)

 	unsigned char* gpu_data;
	hipMalloc((void**)& gpu_data, size_input);
	hipMemcpy(gpu_data, image, size_input, hipMemcpyHostToDevice);

	unsigned char* gpu_data2;
	hipMalloc((void**)& gpu_data2, size_output);

	int pix_per_thread = pixels_output / threads;





	int gpu_block_count = threads > 1024 ? 2 : 1;
	int threads_per_block = threads > 1024 ? (threads / 2) : threads;


	//unsigned char* image, unsigned char* image_out, in, int threads,
	//int width, int height, int width_output, int height_output, int pix_per_thread) {
	addKernel<<<gpu_block_count, threads_per_block >>> (gpu_data, gpu_data2, threads, width, height, width_output, height_output, pix_per_thread);
	hipDeviceSynchronize();

	free(image);
	
	new_image = (unsigned char*)malloc(sizeof(unsigned char) * size_output);
	hipMemcpy(new_image, gpu_data2, size_output, hipMemcpyDeviceToHost);



	exportPNG(output_filename, width_output, height_output, new_image);

}


int main(int argc, char* argv[])
{
	char* input_filename = argv[1];
	int dimension = 3;
	
	convolution(input_filename, "test_output.png",dimension, 16);
    return 0;
}
