#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "lodepng.c"
#include ""

#include <stdio.h>

void convolution(char* input_filename,int dimension, int threads);

void loadPNG(char* input_filename, unsigned int& width, unsigned int& height, unsigned char** output_data)
{
	unsigned error;

	error = lodepng_decode32_file(output_data, &width, &height, input_filename);
	if (error) printf("error %u: %s\n", error, lodepng_error_text(error));
}

void exportPNG(char* output_filename, unsigned int width, unsigned int height,
	unsigned char* data)
{
	lodepng_encode32_file(output_filename, data, width, height);
}

__global__ void addKernel()
{
  
}

void convolution(char* input_filename, int dimension, int threads) {
	unsigned char* image = nullptr;
	unsigned int width, height;
	loadPNG(input_filename, width, height, &image);
	int size = width * height * 4;


	int mul_op_width = width - dimension + 1;
	int mul_op_height = height - dimension + 1;
	int total_mul_op = mul_op_width*mul_op_height; // Number of Matrix Multiplication to be performed;




	return;
}

int main()
{
    
	char* input_filename = "input.png";
	int dimension = 3; //3,5 or 7
	int threads = 1;

	convolution(input_filename,dimension,threads);

   
    return 0;
}

